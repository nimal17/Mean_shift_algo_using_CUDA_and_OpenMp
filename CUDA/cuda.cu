#include "hip/hip_runtime.h"
#include <chrono>
#include "constants.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "utils.h"

namespace mean_shift::cuda {

    __global__ void performMeanShift(float *points, float *updatedPoints) {
        size_t threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (threadId < N) {
            size_t currentRow = threadId * D;
            float newPosition[D] = {0.};
            float totalWeight = 0.;
            for (size_t i = 0; i < N; ++i) {
                size_t otherRow = i * D;
                float squaredDistance = 0.;
                for (size_t j = 0; j < D; ++j) {
                    float distanceComponent = points[currentRow + j] - points[otherRow + j];
                    squaredDistance += distanceComponent * distanceComponent;
                }
                if (squaredDistance <= radius) {
                    float weight = expf(-squaredDistance / sigma2Sqr);
                    for (size_t j = 0; j < D; ++j) {
                        newPosition[j] += weight * points[otherRow + j];
                    }
                    totalWeight += weight;
                }
            }
            for (size_t j = 0; j < D; ++j) {
                updatedPoints[currentRow + j] = newPosition[j] / totalWeight;
            }
        }
    }

}

int main() {

    constexpr auto numPoints = mean_shift::cuda::N;
    constexpr auto dimensions = mean_shift::cuda::D;
    constexpr auto numThreads = mean_shift::cuda::threads;
    constexpr auto numBlocks = mean_shift::cuda::blocks;
    const auto dataFilePath = mean_shift::cuda::filePath; 
    
    std::array<float, numPoints * dimensions> currentData = mean_shift::cuda::utils::get_file<numPoints, dimensions>(dataFilePath, ',');
    std::array<float, numPoints * dimensions> nextData {};
    float *deviceCurrentData;
    float *deviceNextData;

    size_t bytesRequired = numPoints * dimensions * sizeof(float);
    hipMalloc(&deviceCurrentData, bytesRequired);
    hipMalloc(&deviceNextData, bytesRequired);


    hipMemcpy(deviceCurrentData, currentData.data(), bytesRequired, hipMemcpyHostToDevice);
    hipMemcpy(deviceNextData, nextData.data(), bytesRequired, hipMemcpyHostToDevice);

    auto startTime = std::chrono::system_clock::now();
    for (size_t iteration = 0; iteration < mean_shift::cuda::numIterations; ++iteration) {
        mean_shift::cuda::performMeanShift<<<numBlocks, numThreads>>>(deviceCurrentData, deviceNextData);
        hipDeviceSynchronize();
        mean_shift::cuda::utils::swap_func(deviceCurrentData, deviceNextData);
    }
    hipMemcpy(currentData.data(), deviceCurrentData, bytesRequired, hipMemcpyDeviceToHost);
    auto detectedCentroids = mean_shift::cuda::utils::conv_centroid<numPoints, dimensions>(currentData, mean_shift::cuda::minDist);
    auto endTime = std::chrono::system_clock::now();
    auto durationMS = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
    std::cout << "\nTime taken : " << durationMS << " ms\n" << std::endl;

    hipFree(deviceCurrentData);
    hipFree(deviceNextData);

    mean_shift::cuda::utils::print_output<dimensions>(detectedCentroids);
    std::cout << "There are " << detectedCentroids.size() << " centroids.\n";

    

    return 0;
}
